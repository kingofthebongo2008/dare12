#include "hip/hip_runtime.h"
#include "precompiled.h"
#include <cstdint>
#include <memory>
#include <algorithm>

#include "imaging_utils.h"
#include "cuda_imaging.h"


namespace freeform
{

    static inline std::tuple < dim3, dim3 > create_texture_kernel_params( uint32_t width, uint32_t height )
    {
        //1x1 squares
        const dim3 work_items(width, height, 1);
        const dim3 per_block(16, 16, 1);

        const dim3 grid((work_items.x + per_block.x - 1) / per_block.x, (work_items.y + per_block.y - 1) / per_block.y);
        return std::make_tuple(grid, per_block);
    }

    __device__ static inline uint8_t    compute_sobel(
        float   ul, // upper left
        float  um, // upper middle
        float  ur, // upper right
        float  ml, // middle left
        float  mm, // middle (unused)
        float  mr, // middle right
        float  ll, // lower left
        float  lm, // lower middle
        float  lr  // lower right
        )
    {
        float   horizontal  = ur + 2 * mr + lr - ul - 2 * ml - ll;
        float   vertical    = ul + 2 * um + ur - ll - 2 * lm - lr;

        float   gradient = sqrt(horizontal * horizontal + vertical * vertical);

        return  gradient;
    }


    static __global__ void sobel(const uint8_t* img_in, uint8_t* img_out, cuda::image_kernel_info src, cuda::image_kernel_info  dst)
    {
        using namespace cuda;
        auto x = blockIdx.x * blockDim.x + threadIdx.x;
        auto y = blockIdx.y * blockDim.y + threadIdx.y;

        if (is_in_interior(src, x, y))
        {
            const uint8_t* pix00 = sample_2d< uint8_t, border_type::clamp >(img_in, src, x - 1, y - 1 );
            const uint8_t* pix01 = sample_2d< uint8_t, border_type::clamp >(img_in, src, x - 0, y - 1);
            const uint8_t* pix02 = sample_2d< uint8_t, border_type::clamp >(img_in, src, x + 1, y - 1);


            const uint8_t* pix10 = sample_2d< uint8_t, border_type::clamp> (img_in, src, x - 1, y);
            const uint8_t* pix11 = sample_2d< uint8_t, border_type::clamp> (img_in, src, x - 0, y);
            const uint8_t* pix12 = sample_2d< uint8_t, border_type::clamp> (img_in, src, x + 1, y);

            const uint8_t* pix20 = sample_2d< uint8_t, border_type::clamp >(img_in, src, x - 1, y + 1);
            const uint8_t* pix21 = sample_2d< uint8_t, border_type::clamp >(img_in, src, x - 0, y + 1);
            const uint8_t* pix22 = sample_2d< uint8_t, border_type::clamp >(img_in, src, x + 1, y + 1);

            auto  u00 = *pix00 / 255.0f;
            auto  u01 = *pix01 / 255.0f;
            auto  u02 = *pix02 / 255.0f;

            auto  u10 = *pix10 / 255.0f;
            auto  u11 = *pix11 / 255.0f;
            auto  u12 = *pix12 / 255.0f;

            auto  u20 = *pix20 / 255.0f;
            auto  u21 = *pix21 / 255.0f;
            auto  u22 = *pix22 / 255.0f;


            auto  r = compute_sobel(
                u00, u01, u02,
                u10, u11, u12,
                u20, u21, u22
                );
            
            write_2d<float>(img_out, dst, x, y, r );
        }
    }

    imaging::cuda_texture create_canny_texture(const imaging::cuda_texture& texture_grayscale, float threshold)
    {
        using namespace cuda;
        auto width = texture_grayscale.get_width();
        auto height = texture_grayscale.get_height();
        auto t = cuda::create_cuda_texture<imaging::image_type::float32>(width, height);

        auto params     = create_texture_kernel_params(width, height);

        sobel << < std::get<0>(params), std::get<1>(params) >> >  (texture_grayscale.get_gpu_pixels(), t.get_gpu_pixels(), create_image_kernel_info(texture_grayscale), create_image_kernel_info(t));

        cuda::throw_if_failed(hipDeviceSynchronize());

        return std::move(t);
    }
}


