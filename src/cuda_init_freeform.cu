#include "hip/hip_runtime.h"
#include "precompiled.h"

#include "freeform_patch.h"

#include <thrust/transform.h>

#include "math_vector.h"



namespace freeform
{
    __device__ patch cub_bezier_interpol(patch p)
    {

        float v00 = 1.0f;
        float v01 = 0.0f;
        float v02 = 0.0f;
        float v03 = 0.0f;

        float v10 = -5.0f / 6.0f;
        float v11 = 3.0f;
        float v12 = -3.0f / 2.0f;
        float v13 = 1.0f  / 3.0f;

        float v20 = 1.0f / 3.0f;
        float v21 = -3.0f / 2.0f;
        float v22 = 3.0f;
        float v23 = -5.0f / 6.0f;

        float v30 = 0.0f;
        float v31 = 0.0f;
        float v32 = 0.0f;
        float v33 = 1.0f;





    }



    struct generate_patch
    {
        float m_center_x;
        float m_center_y;
        float m_radius;
        float m_step;

        generate_patch(float center_x, float center_y, float radius, float step) :
            m_center_x(center_x)
            , m_center_y(center_y)
            , m_radius(radius)
            , m_step(step)
        {

        }

        __device__ float x(uint32_t i, uint32_t step) const
        {
            return m_center_x + m_radius * cosf((i + step) * m_step);
        }

        __device__ float y(uint32_t i, uint32_t step) const
        {
            return m_center_y + m_radius * sinf((i + step) * m_step);
        }

        __device__ thrust::tuple< freeform::patch, freeform::patch > operator() (uint32_t i) const
        {
            float x0 = x(3 * i, 0);
            float x1 = x(3 * i, 1);
            float x2 = x(3 * i, 2);
            float x3 = x(3 * i, 3);

            float y0 = y(3 * i, 0);
            float y1 = y(3 * i, 1);
            float y2 = y(3 * i, 2);
            float y3 = y(3 * i, 3);


            freeform::patch p0 = { x0, x1, x2, x3, y0, y1, y2, y3 };
            freeform::patch p1 = { x0+100, x1+100, x2+100, x3+100, y0+100, y1+100, y2+100, y3+100 };

            return thrust::make_tuple ( p0, p1 );
        }
    };

    void inititialize_free_form( uint32_t center_image_x, uint32_t center_image_y, float radius, uint32_t patch_count )
    {
        thrust::device_vector<float> x;
        thrust::device_vector<float> y;

        thrust::device_vector<freeform::patch> patches;
        thrust::device_vector<freeform::patch> patches_n;

        auto pi = 3.1415926535f;
        auto pas = 2 * pi / patch_count;
        auto pas_pt_patch = pas / 3.0f;

        auto iterations = static_cast<uint32_t> (ceilf(2 * pi / pas_pt_patch));

    
        patches.resize( iterations / 3 );
        patches_n.resize(iterations / 3);


        auto begin  = thrust::make_counting_iterator(0);
        auto end    = begin + iterations / 3;
        auto o      = thrust::make_zip_iterator(thrust::make_tuple(patches.begin(), patches_n.begin()));

        thrust::transform(begin, end, o, generate_patch(static_cast<float> (center_image_x), static_cast<float> (center_image_y), radius, pas_pt_patch));


        thrust::host_vector<freeform::patch> r;

        r.resize(iterations / 3 );

        thrust::copy(patches_n.begin(), patches_n.end(), r.begin());
        thrust::copy(r.begin(), r.end(), std::ostream_iterator< freeform::patch >(std::cout, " "));
    }
}


